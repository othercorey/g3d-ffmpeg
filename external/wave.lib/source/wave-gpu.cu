#include "hip/hip_runtime.h"
/*
Copyright 2018, NVIDIA

2-Clause BSD License (https://opensource.org/licenses/BSD-2-Clause)

Redistribution and use in source and binary forms, with or without modification, are permitted
provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this list of conditions
and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice, this list of
conditions and the following disclaimer in the documentation and/or other materials provided
with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR
TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#define NOMINMAX
#define _USE_MATH_DEFINES
#define _NVISA
#define NVVM_ADDRESS_SPACE_0_GENERIC
#define __STDC_LIMIT_MACROS
#define __STDC_CONSTANT_MACROS
#define INTERNAL_DEVELOPER
#define DEVELOP 1
#define _VARIADIC_MAX 10
#define CUDA_64_BIT_DEVICE_CODE

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
//#include <hip/hip_fp16.h>

#define SAMPLE(id, x, y, layer, lod) rtTex2DLod<float4>(id, x, y, lod)
// OptiX programming model in brief:
//
// The INTERSECT callback is determined by the geometry (we use one: triangle). It only
// has access to variables bound on the geometry by the host, not materials. So it can't
// perform the alpha test directly.
// 
// The CLOSEST-HIT and ANY-HIT callbacks are determined by the material returned from the
// intersect callback and the ray type specified in the generator. They have access to
// the material variables. We use any hit to perform alpha test. For occlusion rays, the closest
// hit does nothing. For regular rays, closest hit performs full material sampling.
//
// For our application, all rays in a wavefront are either occlusion or regular cast; there is
// no mixing of ray types within a call.

using namespace optix;

/** float4 -> float3 */
inline __device__ float3 xyz(float4 v) { return make_float3(v.x, v.y, v.z); }

rtBuffer<float4, 2>         rayOriginBuffer;
rtBuffer<float4, 2>         rayDirectionBuffer;

rtDeclareVariable(rtObject,     root, , );
rtDeclareVariable(uint2,        launch_index, rtLaunchIndex, );
// rtDeclareVariable(uint2,        launch_dim,   rtLaunchDim, );

// OptiX doesn't support bool, so we use ints here
rtDeclareVariable(int,          backfaceCull, , );
rtDeclareVariable(int,          alphaTest, , );
rtDeclareVariable(int,          lod, , );
rtDeclareVariable(unsigned int, visibilityMask, , );
rtDeclareVariable(float,        partialCoverageThreshold, , );

// For occlusion cast
rtBuffer<unsigned char, 2>      hitOutBuffer;  

// For regular cast
rtBuffer<uchar4, 2>             material0OutBuffer;  
rtBuffer<uchar4, 2>             material1OutBuffer;  
rtBuffer<uchar4, 2>             material2OutBuffer;  
rtBuffer<float4, 2>             material3OutBuffer;  

rtBuffer<uint4, 2>              hitLocationOutBuffer;
rtBuffer<float4, 2>             shadingNormalOutBuffer;
rtBuffer<float4, 2>             positionOutBuffer;


rtBuffer<float, 2>              rayConeAnglesBuffer;

// Information passed between the callbacks
struct Hit {
    // The following are computed in intersect:
    //float           t;
    int              primitiveIndex;
    //int             materialIndex;
    float2           barycentrics;

    //
    //unsigned int    backface;

     //These are non-unit until the closest hit:
    //float3          geometricNormal;
    //float3          interpolatedNormal;
    //float3          interpolatedTangent;
    //
    //// The following are computed in closest-hit
    //// for regular casts:
    //
    ///** World-space shading normal, bump value */
    //float4          texN;
    //float4          tex0;
    //float4          tex1;
    //float4          tex2;
    //float4          tex3;
    //float           constant;
    //unsigned int    flags;
};


// Geometry
rtBuffer<float3> vertexPosition;
rtBuffer<float2> vertexTexcoord;
rtBuffer<float3> vertexNormal;
rtBuffer<float4> vertexTangent;
rtBuffer<int3>   geometryIndex;

rtDeclareVariable(int, twoSided, , );


// Variables passed between functions. Not visible on the host
rtDeclareVariable(Hit, hit_attr, attribute hit_attr, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


// Triangle intersection with optional backface culling.
// Intersect runs at a time when we do not yet know if this is going to
// be the first hit (or any hit at all!).
// In the triangle API, this is an *attributes* program, not an intersection program.
RT_PROGRAM void intersect() {
    hit_attr.primitiveIndex = rtGetPrimitiveIndex();
    hit_attr.barycentrics = rtGetTriangleBarycentrics();
}


// Materials
//
// These are material properties that are automatically bound by OptiX
// when the material's corresponding any_hit/closest_hit shader is invoked

//rtTextureSampler<float4, 2> texNSampler;
//rtTextureSampler<float4, 2> tex0Sampler;
//rtTextureSampler<float4, 2> tex1Sampler;
//rtTextureSampler<float4, 2> tex2Sampler;
//rtTextureSampler<float4, 2> tex3Sampler;

rtDeclareVariable(int, texNSampler, , );
rtDeclareVariable(int, tex0Sampler, , );
rtDeclareVariable(int, tex1Sampler, , );
rtDeclareVariable(int, tex2Sampler, , );
rtDeclareVariable(int, tex3Sampler, , );

rtDeclareVariable(float4, texNScale, , );
rtDeclareVariable(float4, texNBias, , );

rtDeclareVariable(float4, tex0Scale, , );
rtDeclareVariable(float4, tex0Bias, , );

rtDeclareVariable(float4, tex1Scale, , );
rtDeclareVariable(float4, tex1Bias, , );

rtDeclareVariable(float4, tex2Scale, , );
rtDeclareVariable(float4, tex2Bias, , );

rtDeclareVariable(float4, tex3Scale, , );
rtDeclareVariable(float4, tex3Bias, , );

rtDeclareVariable(int,    flags, , );
rtDeclareVariable(float,  constant, , );

rtDeclareVariable(Hit,    hit_prd, rtPayload, );


static __device__ __inline__ optix::uchar4 make_color(const optix::float4& c)
{
    return optix::make_uchar4(static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* B */
                              static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
                              static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* R */
                              static_cast<unsigned char>(__saturatef(c.w)*255.99f)); /* A */
}

RT_PROGRAM void closestHit() {

    //// Vertex indices
    const int3 v_idx = geometryIndex[hit_attr.primitiveIndex];

    const float3 p0 = vertexPosition[v_idx.x];
    const float3 p1 = vertexPosition[v_idx.y];
    const float3 p2 = vertexPosition[v_idx.z];

    //// Ray-triangle intersection
    //const float3 e0 = p1 - p0;
    //const float3 e1 = p0 - p2;
    //const float3 geometricNormal = cross(e1, e0);

    //const float3 wsP0 = rtTransformPoint(RT_OBJECT_TO_WORLD, p0);
    //const float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));
    //// Compute t
    //const float d = dot(wsGeometricNormal, ray.direction);
    //const float3 e2 = (1.0f / d) * (wsP0 - ray.origin);
    //const float t = dot(wsGeometricNormal, e2);

    //hit_attr.backface = (d > 0.0f) ? 1 : 0;
    const float beta = hit_attr.barycentrics.x;
    const float gamma = hit_attr.barycentrics.y;

    hit_prd = hit_attr;


    const float3 hitPos = rtTransformPoint(RT_OBJECT_TO_WORLD, p1 * beta + p2 * gamma + p0 * (1.0f - beta - gamma));


    // We've encoded the desired MIP level in the ray direction length
    int mipLevel = lod; // 0 if not set
        //(lod == 0) ? length(ray.direction) - 1 : lod;

    float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, vertexNormal[v_idx.y] * beta + vertexNormal[v_idx.z] * gamma + vertexNormal[v_idx.x] * (1.0f - beta - gamma)));

    // Ray cones, empty (set to 0.0f) if not set.
    float angle = rayConeAnglesBuffer[launch_index];
    mipLevel += log(angle * length(hitPos - ray.origin) * (1.0f / max(abs(dot(normal, xyz(rayDirectionBuffer[launch_index]))), 0.000001f)));

    const float frontface = (twoSided || rtIsTriangleHitFrontFace()) ? 1.0f : 0.0f;
    const float flip = (twoSided && ! rtIsTriangleHitFrontFace()) ? 0.0f : 1.0f;


    // Old wave.lib returned hit distance in the w channel as an idiosyncrasy of 
    // the DDGI project. We now instead return 1.0f to conform to the G3D::TriTree API.
    positionOutBuffer[launch_index] = make_float4(hitPos, 1.0f);

    const float2 texcoord = vertexTexcoord[v_idx.y] * beta + vertexTexcoord[v_idx.z] * gamma + vertexTexcoord[v_idx.x] * (1.0f - beta - gamma);

    const float4 texNSample = SAMPLE(texNSampler, texcoord.x, texcoord.y, 0, mipLevel) * texNScale + texNBias;

    material0OutBuffer[launch_index] = make_color(SAMPLE(tex0Sampler, texcoord.x, texcoord.y, 0, mipLevel) * tex0Scale + tex0Bias);
    material1OutBuffer[launch_index] = make_color(SAMPLE(tex1Sampler, texcoord.x, texcoord.y, 0, mipLevel) * tex1Scale + tex1Bias);
    material2OutBuffer[launch_index] = make_color(SAMPLE(tex2Sampler, texcoord.x, texcoord.y, 0, mipLevel) * tex2Scale + tex2Bias);
    material3OutBuffer[launch_index] = make_float4(xyz(SAMPLE(tex3Sampler, texcoord.x, texcoord.y, 0, mipLevel) * tex3Scale + tex3Bias), constant);
    
    // Normal is automatically flipped if we hit a "backface" on a twoSided object. It is *not* flipped if we hit a backface otherwise.
    shadingNormalOutBuffer[launch_index] = make_float4(normal * (2.0f * flip - 1.0f), frontface);
    
     //TODO: Morgan, figure out the coordinate system for this section
     //Transform the texn.xyz value to world space using the tangent frame
    //const float3 tangent2 = normalize(cross(hit_prd.interpolatedTangent, hit_prd.interpolatedNormal));
    //const float3 tsNormal = xyz(hit_prd.texN);
    //hit_prd.texN.x = dot(hit_prd.interpolatedTangent, tsNormal);
    //hit_prd.texN.y = dot(tangent2, tsNormal);
    //hit_prd.texN.z = dot(hit_prd.interpolatedNormal, tsNormal);
}


RT_PROGRAM void occlusionClosestHit() {
    hit_prd = hit_attr;
}


// Used for both occlusion cast and regular cast
RT_PROGRAM void anyHit() {
//// Vertex indices
    const int3 v_idx = geometryIndex[hit_attr.primitiveIndex];

    const float beta = hit_attr.barycentrics.x;
    const float gamma = hit_attr.barycentrics.y;
    const int mipLevel = lod;

    const float2 texcoord = vertexTexcoord[v_idx.y] * beta + vertexTexcoord[v_idx.z] * gamma + vertexTexcoord[v_idx.x] * (1.0f - beta - gamma);
    //printf("anyHit, index %u, %u\nval = %f\n", launch_index.x, launch_index.y, (rtTex2D<float4>(tex0Sampler, hit_attr.texcoord.x, hit_attr.texcoord.y).w * tex0Scale.w + tex0Bias.w));
    float test = SAMPLE(tex0Sampler, texcoord.x,texcoord.y, 0, mipLevel).w * tex0Scale.w + tex0Bias.w;
    if ( alphaTest && ( (test < partialCoverageThreshold) || (test == 0.0f) )) {
        // Failed backface test
        rtIgnoreIntersection();
    }

}

//const float INFINITY = __int_as_float(0x7f800000);

/** Shared by both cast() and occlusionCast() */
__device__ void trace(Hit& hit_prd, int rayType) {
    //printf("trace, index %u, %u\n", launch_index.x, launch_index.y);
    //if (launch_index.x > (uint)32 || launch_index.y > (uint)32) {
    //    printf("Out! %u, %u\n", launch_index.x, launch_index.y);
    //}
    //hit_prd.t           = -1.0f;

    hit_prd.primitiveIndex = -1;

    //const float scale = 3.0f;
    //const float x = scale * (-0.5f + ((float)launch_index.x / 1920.0f));
    //const float y = scale * (-0.5f + ((float)launch_index.y / 1080.0f));
    const float4 direction = rayDirectionBuffer[launch_index];
	// Early out, faster than degenerate rays according to Adam.
	//if (direction.w == -1.0f) {
	//	return;
	//}
    const float4 origin = rayOriginBuffer[launch_index];
        //make_float4(x, y, 0.0f, 0.0f);
    //make_float4(0.0f, 0.0f, -1.0f, 100.0f);//
    //printf("origin, direction = (%f, %f, %f, %f), (%f, %f, %f, %f)\n", origin.x, origin.y, origin.z, origin.w, direction.x, direction.y, direction.z, direction.w);
    //
    
    // If we're not tracing occlusion rays, disable backface culling.
    rtTrace(root, make_Ray(xyz(origin), xyz(direction), rayType, origin.w, direction.w), hit_prd, visibilityMask, (rayType || !backfaceCull) ? RT_RAY_FLAG_NONE : RT_RAY_FLAG_CULL_BACK_FACING_TRIANGLES);
}


/** Ray generation program for general cast. Because it is a generation function, there is no access to the ray struct. Trying to use it will cause a trap error.*/
RT_PROGRAM void cast() {
    Hit hit_prd;
    trace(hit_prd, 0);
}    

 
RT_PROGRAM void occlusionCast() {
    Hit hit_prd;
    trace(hit_prd, 1);

    hitOutBuffer[launch_index] = (hit_prd.primitiveIndex >= 0) ? 255 : 0;
}


// Only used for debugging
RT_PROGRAM void exception() {
    const unsigned int code = rtGetExceptionCode();
}

/** Writes black to the output buffers.*/
RT_PROGRAM void miss() {
    const float4 missVal = make_float4(0.0f, 0.0f, 0.0f, -1.0f);
    material3OutBuffer[launch_index] = missVal;
    shadingNormalOutBuffer[launch_index] = missVal;
    positionOutBuffer[launch_index] = make_float4(0.0f, 0.0f, 0.0f, INFINITY);
}